#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>

__global__
void unifMem_vectorAdd(int* a, int* b, int* c, int n) {
   int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
   if (tid < n){
      c[tid] = a[tid] + b[tid];
   }
}

void vectors_init(int* a, int* b, int n) {
   for (int i = 0; i < n; i++) {
      a[i] = rand() % 10;
      b[i] = rand() % 10;
   }
}

void error_check(int* a , int* b, int* c, int n) {
   for (int i = 0; i < n; i++) {
      assert(c[i] == a[i] + b[i]);
   }
}

int main() {
   int n = 1 << 16;
   int id = hipGetDevice(&id);
   int *a, *b, *c;
   size_t bytes = n * sizeof(int);

   hipMallocManaged(&a, bytes);
   hipMallocManaged(&b, bytes);
   hipMallocManaged(&c, bytes);
   
   vectors_init(a, b, n);

   int BLOCK_SIZE = 256;
   int GRID_SIZE = (int)ceil(n / BLOCK_SIZE);
   
   hipMemPrefetchAsync (a, bytes, id);
   hipMemPrefetchAsync (b, bytes, id);

   unifMem_vectorAdd <<<GRID_SIZE, BLOCK_SIZE>>> (a, b, c, n);
   
   hipDeviceSynchronize();
   
   hipMemPrefetchAsync(c, bytes, hipCpuDeviceId);

   error_check(a, b, c, n);

   printf("SUCCESS\n");

   return 0;
}

