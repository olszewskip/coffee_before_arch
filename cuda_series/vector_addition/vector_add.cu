#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>

__global__
void vectorAdd(int* a, int* b, int* c, int n) {
   int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
   if (tid < n){
      c[tid] = a[tid] + b[tid];
   }
}

void matrix_init(int* a, int n) {
   for (int i = 0; i < n; i++) {
      a[i] = rand() % 100;
   }
}

void error_check(int* a , int* b, int* c, int n) {
   for (int i = 0; i < n; i++) {
      assert(c[i] == a[i] + b[i]);
   }
}

int main() {
   int n = 1 << 16;
   int *h_a, *h_b, *h_c;
   int *d_a, *d_b, *d_c;

   size_t bytes = n * sizeof(int);

   h_a = (int*)malloc(bytes);
   h_b = (int*)malloc(bytes);
   h_c = (int*)malloc(bytes);
   
   hipMalloc(&d_a, bytes);
   hipMalloc(&d_b, bytes);
   hipMalloc(&d_c, bytes);
   
   matrix_init(h_a, n);
   matrix_init(h_b, n);

   hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
   hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

   int NUM_THREADS = 256;
   int NUM_BLOCKS = (int)ceil(n / NUM_THREADS);

   vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, n);

   hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

   error_check(h_a, h_b, h_c, n);

   printf("SUCCESS\n");

   return 0;
}

